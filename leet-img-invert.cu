#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel = (row * width + col)*4;

    image[pixel] = 255 - image[pixel];
    image[pixel+1] = 255 - image[pixel+1];
    image[pixel+2] = 255 - image[pixel+2];
    

}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}
