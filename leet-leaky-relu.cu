#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
 int my_x = blockDim.x * blockIdx.x + threadIdx.x;
 if(my_x >= N){
    return;
 }
 output[my_x] = input[my_x]>0?input[my_x]:(0.01*input[my_x]);
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
