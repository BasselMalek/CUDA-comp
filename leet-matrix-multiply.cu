#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float *A, const float *B,
                                             float *C, int M, int N, int K) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < K) {
        float sum = 0.0f;
        for (int d = 0; d < N; d++) {
            sum += A[row * N + d] * B[d * K + col];
        }
        C[row * K + col] = sum;
    }
}


// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
